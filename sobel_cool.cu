#include "hip/hip_runtime.h"
/* Example sobel code for ECE574 -- Spring 2023 */
/* By Vince Weaver <vincent.weaver@maine.edu> */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <errno.h>
#include <math.h>

#include <jpeglib.h>

#include <hip/hip_runtime.h>

#include <papi.h>

/* Filters */
static int sobel_x_filter[9]={ -1, 0,+1  ,-2, 0,+2  ,-1, 0,+1};
static int sobel_y_filter[9]={ -1,-2,-1  , 0, 0, 0  , 1, 2,+1};

/* Structure describing the image */
struct image_t {
	int xsize;
	int ysize;
	int depth;	/* bytes */
	unsigned char *pixels;
};

#if 0
__global__
void cuda_generic_convolve (int n, char *in, char *out, int *matrix) {

}

__global__
void cuda_combine (int n, unsigned char *sobel_x,
		unsigned char *sobel_y, unsigned char *output) {
}

#endif


/* very inefficient convolve code */
static void *generic_convolve(struct image_t *input,
				struct image_t *output,
				int *filter) {

	int x,y,k,l,d;
	uint32_t color;
	int sum,depth,width;

	int ystart, yend;

	ystart=0;
	yend=input->ysize;

	depth=input->depth;
	width=input->xsize * input->depth;

	if (ystart==0) ystart=1;
	if (yend==input->ysize) yend=input->ysize-1;

	for(d=0;d<3;d++) {
	   for(x=1;x<input->xsize-1;x++) {
	     for(y=ystart;y<yend;y++) {
		sum=0;
		for(k=-1;k<2;k++) {
		   for(l=-1;l<2;l++) {
			color=input->pixels[((y+l)*width)+(x*depth+d+k*depth)];
			sum+=color * filter[(l+1)*3+(k+1)];
		   }
		}

		if (sum<0) sum=0;
		if (sum>255) sum=255;

		output->pixels[(y*width)+x*depth+d]=sum;
	     }
	   }
	}

	return NULL;
}

static int combine(struct image_t *sobel_x,
			struct image_t *sobel_y,
			struct image_t *output) {
	int i;
	int out;

	for(i=0;i<( sobel_x->depth * sobel_x->xsize * sobel_x->ysize );i++) {

		out=sqrt(
			(sobel_x->pixels[i]*sobel_x->pixels[i])+
			(sobel_y->pixels[i]*sobel_y->pixels[i])
			);
		if (out>255) out=255;
		if (out<0) out=0;
		output->pixels[i]=out;
	}

	return 0;
}

static int load_jpeg(char *filename, struct image_t *image) {

	FILE *fff;
	struct jpeg_decompress_struct cinfo;
	struct jpeg_error_mgr jerr;
	JSAMPROW output_data;
	unsigned int scanline_len;
	int scanline_count=0;

	fff=fopen(filename,"rb");
	if (fff==NULL) {
		fprintf(stderr, "Could not load %s: %s\n",
			filename, strerror(errno));
		return -1;
	}

	/* set up jpeg error routines */
	cinfo.err = jpeg_std_error(&jerr);

	/* Initialize cinfo */
	jpeg_create_decompress(&cinfo);

	/* Set input file */
	jpeg_stdio_src(&cinfo, fff);

	/* read header */
	jpeg_read_header(&cinfo, TRUE);

	/* Start decompressor */
	jpeg_start_decompress(&cinfo);

	printf("output_width=%d, output_height=%d, output_components=%d\n",
		cinfo.output_width,
		cinfo.output_height,
		cinfo.output_components);

	image->xsize=cinfo.output_width;
	image->ysize=cinfo.output_height;
	image->depth=cinfo.output_components;

	scanline_len = cinfo.output_width * cinfo.output_components;
	image->pixels=(unsigned char *)malloc(cinfo.output_width * cinfo.output_height * cinfo.output_components);

	while (scanline_count < cinfo.output_height) {
		output_data = (image->pixels + (scanline_count * scanline_len));
		jpeg_read_scanlines(&cinfo, &output_data, 1);
		scanline_count++;
	}

	/* Finish decompressing */
	jpeg_finish_decompress(&cinfo);

	jpeg_destroy_decompress(&cinfo);

	fclose(fff);

	return 0;
}

static int store_jpeg(const char *filename, struct image_t *image) {

	struct jpeg_compress_struct cinfo;
	struct jpeg_error_mgr jerr;
	int quality=90; /* % */
	int i;

	FILE *fff;

	JSAMPROW row_pointer[1];
	int row_stride;

	/* setup error handler */
	cinfo.err = jpeg_std_error(&jerr);

	/* initialize jpeg compression object */
	jpeg_create_compress(&cinfo);

	/* Open file */
	fff = fopen(filename, "wb");
	if (fff==NULL) {
		fprintf(stderr, "can't open %s: %s\n",
			filename,strerror(errno));
		return -1;
	}

	jpeg_stdio_dest(&cinfo, fff);

	/* Set compression parameters */
	cinfo.image_width = image->xsize;
	cinfo.image_height = image->ysize;
	cinfo.input_components = image->depth;
	cinfo.in_color_space = JCS_RGB;
	jpeg_set_defaults(&cinfo);
	jpeg_set_quality(&cinfo, quality, TRUE);

	/* start compressing */
	jpeg_start_compress(&cinfo, TRUE);

	row_stride=image->xsize*image->depth;

	for(i=0;i<image->ysize;i++) {
		row_pointer[0] = & image->pixels[i * row_stride];
		jpeg_write_scanlines(&cinfo, row_pointer, 1);
	}

	/* finish compressing */
	jpeg_finish_compress(&cinfo);

	/* close file */
	fclose(fff);

	/* clean up */
	jpeg_destroy_compress(&cinfo);

	return 0;
}

int main(int argc, char **argv) {

	struct image_t image,sobel_x,sobel_y,new_image;
	long long start_time,load_time;
	long long combine_after=0,combine_before=0;
	long long convolve_after=0,convolve_before=0;
	long long copy_before=0,copy_after=0,copy2_before=0,copy2_after=0;
	long long store_after,store_before;

	/* Check command line usage */
	if (argc<2) {
		fprintf(stderr,"Usage: %s image_file\n",argv[0]);
		return -1;
	}

	PAPI_library_init(PAPI_VER_CURRENT);

	start_time=PAPI_get_real_usec();

	/* Load an image */
	load_jpeg(argv[1],&image);

	load_time=PAPI_get_real_usec();

	/* Allocate space for output image */
	new_image.xsize=image.xsize;
	new_image.ysize=image.ysize;
	new_image.depth=image.depth;
	new_image.pixels=
		(unsigned char *)calloc(image.xsize*image.ysize*image.depth,
					sizeof(char));

	/* Allocate space for output image */
	sobel_x.xsize=image.xsize;
	sobel_x.ysize=image.ysize;
	sobel_x.depth=image.depth;
	sobel_x.pixels=
		(unsigned char *)calloc(image.xsize*image.ysize*image.depth,
					sizeof(char));

	/* Allocate space for output image */
	sobel_y.xsize=image.xsize;
	sobel_y.ysize=image.ysize;
	sobel_y.depth=image.depth;
	sobel_y.pixels=
		(unsigned char *)calloc(image.xsize*image.ysize*image.depth,
					sizeof(char));

	convolve_before=PAPI_get_real_usec();

	/* sobel x convolution */
	generic_convolve(&image,&sobel_x,sobel_x_filter);

	/* sobel y convolution */
	generic_convolve(&image,&sobel_y,sobel_y_filter);

	convolve_after=PAPI_get_real_usec();

	/* Combine to form output */

	combine_before=PAPI_get_real_usec();

	combine(&sobel_x,&sobel_y,&new_image);

	/* REPLACE THE ABOVE WITH YOUR CODE */
	/* IT SHOULD ALLOCATE SPACE ON DEVICE */
	/* COPY SOBEL_X and SOBEL_Y data to device */
	/* RUN THE KERNEL */
	/* THEN COPY THE RESULTS BACK */

	combine_after=PAPI_get_real_usec();

	store_before=PAPI_get_real_usec();

	/* Write data back out to disk */
	store_jpeg("out.jpg",&new_image);

	store_after=PAPI_get_real_usec();

	/* Print timing results */
	printf("Load time: %lld\n",load_time-start_time);
        printf("Convolve time: %lld\n",convolve_after-convolve_before);
	printf("Copy host to device: %lld\n",(copy_after-copy_before));
        printf("Combine time: %lld\n",combine_after-combine_before);
	printf("Copy device to host: %lld\n",(copy2_after-copy2_before));
        printf("Store time: %lld\n",store_after-store_before);
	printf("Total time = %lld\n",store_after-start_time);

	return 0;
}
